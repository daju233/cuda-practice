
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cmath>
#define TILE_WIDTH 32 // 最简单的情况，瓦片的维度等于块的维度

__global__ void MatrixMulKernel(float *M, float *N, float *P, int width)
{
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;//共享内存数组和TILE_WIDTH显然不能超过32

    // 标记当前元素
    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;

    // 遍历width/TILE_WIDTH次，这样可以将需要的元素都加载好到shared memory中
    if ((Row < width) && (Col < width))
    {
        float Pvalue = 0;
        // ph=tilecount
        for (int ph = 0; ph < width / TILE_WIDTH; ++ph)
        {

            // 加载到shared memory中
            Mds[ty][tx] = M[Row * width + ph * TILE_WIDTH + tx];
            Nds[ty][tx] = N[(ph * TILE_WIDTH + ty) * width + Col];
            __syncthreads();
            for (int k = 0; k < TILE_WIDTH; ++k)
            {
                Pvalue += Mds[ty][k] * Nds[k][tx];
            }
            __syncthreads();
        }
        P[Row * width + Col] = Pvalue;
    }
}

int main()
{
    const int width = TILE_WIDTH;
    float *M_d, *N_d, *P_d;

    size_t size = width * width * sizeof(float);

    std::vector<float> h_Matrix_one(width * width, 4.0f);
    std::vector<float> h_Matrix_two(width * width, 2.0f);
    std::vector<float> h_Res(width * width, 0.0f);

    hipMalloc((void **)&M_d, size);
    hipMalloc((void **)&N_d, size);
    hipMalloc((void **)&P_d, size);

    hipMemcpy(M_d, h_Matrix_one.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(N_d, h_Matrix_two.data(), size, hipMemcpyHostToDevice);

    dim3 dimBlock(32, 32, 1);      // 每个块1024个线程 //为什么要这么多？？？
    dim3 dimGrid(4, 4, 1); // 创建一个width*width个块的网格

    MatrixMulKernel<<<dimGrid, dimBlock>>>(M_d, N_d, P_d, width);

    hipMemcpy(h_Res.data(), P_d, size, hipMemcpyDeviceToHost);

    hipFree(M_d);
    hipFree(N_d);
    hipFree(P_d);
    // std::cout << "Matrix C (A * B):" << std::endl;
    // for (int i = 0; i < width; ++i) {
    //     for (int j = 0; j < width; ++j) {
    //         std::cout << h_Res[i * width + j] << "\t";
    //     }
    //     std::cout << std::endl;
    // }
}