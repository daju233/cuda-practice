#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <iomanip>
#include <cmath>

__global__ void MatrixMulKernel(float *M, float *N, float *P, int width)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if ((row < width) && (col < width))
    {
        float Pvalue = 0;
        for (int k = 0; k < width; ++k)
        {
            Pvalue += M[row * width + k] * N[k * width + col];
        }
        P[row * width + col] = Pvalue;
    }
}

int main()
{
    const int width = 4;
    size_t size = width * width * sizeof(float);
    std::vector<float> h_M(width * width);
    std::vector<float> h_N(width * width);
    std::vector<float> h_P(width * width, 0.0f);
    std::vector<float> h_ref(width * width, 0.0f);

    float *d_M, *d_N, *d_P;
    hipMalloc()
}