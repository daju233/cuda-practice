
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cmath>

__global__ void MatrixMulKernel(float *M, float *N, float *P, int width)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if ((row < width) && (col < width))//row col = 0,1
    {
        float Pvalue = 0;
        for (int k = 0; k < width; ++k)//k<=2
        {
            Pvalue += M[row * width + k] * N[k * width + col];
        }        
        P[row * width + col] = Pvalue;
    }
}

int main()
{
    const int width = 2;
    float *M_d, *N_d, *P_d;

    size_t size = width * width * sizeof(float);

    std::vector<float> h_Matrix_one(width * width, 4.0f);
    std::vector<float> h_Matrix_two(width * width, 2.0f);
    std::vector<float> h_Res(width * width, 0.0f);

    hipMalloc((void **)&M_d, size);
    hipMalloc((void **)&N_d, size);
    hipMalloc((void **)&P_d, size);

    hipMemcpy(M_d, h_Matrix_one.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(N_d, h_Matrix_two.data(), size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(2, 2);
    dim3 numBlocks(1, 1);
    //这样写不行 MatrixMulKernel<<<1,2,2>>>(M_d,N_d,P_d,width);
    MatrixMulKernel<<<numBlocks,threadsPerBlock>>>(M_d,N_d,P_d,width);//这样可以，为什么

    hipMemcpy(h_Res.data(), P_d, size, hipMemcpyDeviceToHost);

    hipFree(M_d);
    hipFree(N_d);
    hipFree(P_d);
    for(float elem:h_Res){
        printf("%f\n",elem);
    }
}