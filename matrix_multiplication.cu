
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cmath>

__global__ void MatrixMulKernel(float *M, float *N, float *P, int width)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if ((row < width) && (col < width)) // row col = 0,1
    {
        float Pvalue = 0;
        for (int k = 0; k < width; ++k) // k<=2
        {
            Pvalue += M[row * width + k] * N[k * width + col];
        }
        P[row * width + col] = Pvalue;
    }
}

int main()
{
    const int width = 2;
    float *M_d, *N_d, *P_d;

    size_t size = width * width * sizeof(float);

    std::vector<float> h_Matrix_one={4.0f,4.0f,2.0f,2.0f};
    std::vector<float> h_Matrix_two(width * width, 2.0f);
    std::vector<float> h_Res(width * width, 0.0f);

    hipMalloc((void **)&M_d, size);
    hipMalloc((void **)&N_d, size);
    hipMalloc((void **)&P_d, size);

    hipMemcpy(M_d, h_Matrix_one.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(N_d, h_Matrix_two.data(), size, hipMemcpyHostToDevice);

    dim3 dimGrid(1, 1, 1);
    dim3 dimBlock(2, 2, 1);
    // 这样写不行 MatrixMulKernel<<<1,2,2>>>(M_d,N_d,P_d,width);
    MatrixMulKernel<<<dimGrid, dimBlock>>>(M_d, N_d, P_d, width); // 这样可以，为什么

    hipMemcpy(h_Res.data(), P_d, size, hipMemcpyDeviceToHost);

    hipFree(M_d);
    hipFree(N_d);
    hipFree(P_d);
    for (float elem : h_Res)
    {
        printf("%f\n", elem);
    }
}