#include <hip/hip_runtime.h>
#include<stdio.h>
#include <vector>

__global__ void vecAddKernel(float *A, float *B, float *C, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        C[i] = A[i] + B[i];
    }
}

int main()
{
    float *A_d, *B_d, *C_d;
    float size = 114514 * sizeof(float);

    std::vector<float> A(114514,1.0f);   
    std::vector<float> B(114514,1.0f);  
    std::vector<float> C(114514,0.0f);  

    hipMalloc((void **)&A_d, size);
    hipMalloc((void **)&B_d, size);
    hipMalloc((void **)&C_d, size);

    hipMemcpy(A_d, A.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B.data(), size, hipMemcpyHostToDevice);
    // 第一个配置参数给出了网格中的块数，第二个指定了每个块中的线程数。在这个例子中，每个块中有256个线程。
    //是否有更好的写法？
    vecAddKernel<<<ceil(114514 / 256.0), 256.0>>>(A_d, B_d, C_d, size);
    hipMemcpy(C.data(),C_d,size,hipMemcpyDeviceToHost);
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
    printf("%f",C[123]);
}