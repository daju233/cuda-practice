#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void vecAddKernel(float *A, float *B, float *C, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        C[i] = A[i] + B[i];
    }
}

int main()
{
    float *A_d, *B_d, *C_d;
    float size = 114514 * sizeof(float);
    float A[114514]={1,1,1,1};
    float B[114514]={1,1,1,1};
    float C[114514]={0,0,0,0};
    hipMalloc((void **)&A_d, size);
    hipMalloc((void **)&B_d, size);
    hipMalloc((void **)&C_d, size);
    hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, size, hipMemcpyHostToDevice);
    // 第一个配置参数给出了网格中的块数，第二个指定了每个块中的线程数。在这个例子中，每个块中有256个线程。
    vecAddKernel<<<ceil(114514 / 256.0), 256.0>>>(A_d, B_d, C_d, size);
    hipMemcpy(C,C_d,size,hipMemcpyDeviceToHost);
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
    printf("%f",C[1]);
}