
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cmath>

__global__ void MatrixMulKernel(float *M, float *N, float *P, int width)
{
    // 每个线程负责一个输出元素
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if ((row < width) && (col < width)) // row col < width
    {
        float Pvalue = 0;
        for (int k = 0; k < width; ++k) // k < width
        {
            Pvalue += M[row * width + k] * N[k * width + col]; // 一整行和一整列
        }
        P[row * width + col] = Pvalue;
    }
}

int main()
{
    const int width = 32;
    float *M_d, *N_d, *P_d;

    size_t size = width * width * sizeof(float);

    std::vector<float> h_Matrix_one(width * width, 4.0f);
    std::vector<float> h_Matrix_two(width * width, 2.0f);
    std::vector<float> h_Res(width * width, 0.0f);

    hipMalloc((void **)&M_d, size);
    hipMalloc((void **)&N_d, size);
    hipMalloc((void **)&P_d, size);

    hipMemcpy(M_d, h_Matrix_one.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(N_d, h_Matrix_two.data(), size, hipMemcpyHostToDevice);

    dim3 dimBlock(16,16,1);//每个块256个线程
    dim3 dimGrid(width,width,1);//创建一个width*width个块的网格

    MatrixMulKernel<<<dimGrid, dimBlock>>>(M_d, N_d, P_d, width);

    hipMemcpy(h_Res.data(), P_d, size, hipMemcpyDeviceToHost);

    hipFree(M_d);
    hipFree(N_d);
    hipFree(P_d);
    // std::cout << "Matrix C (A * B):" << std::endl;
    // for (int i = 0; i < width; ++i) {
    //     for (int j = 0; j < width; ++j) {
    //         std::cout << h_Res[i * width + j] << "\t";
    //     }
    //     std::cout << std::endl;
    // }
}